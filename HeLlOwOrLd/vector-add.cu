#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x + blockInx.x * blockDim.x;
    c[index] = a[index] + b[index];
}
int main() {
    int *a, *b, *c;
    
    // host copies of variables a, b & c
    int *d_a, *d_b, *d_c;
    
    // device copies of variables a, b & c
    int size = N * sizeof(int);
    
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    // Setup input values  
    a = (int *) malloc(size); 
    b = (int *) malloc(size);
    c = (int *) malloc(size);
    
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = -i;
    }
    
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    
    // Launch add() kernel on GPU
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    
    if(err!=hipSuccess) {
        printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }
    
    for (int i = 0; i < 10; i++) {
        printf("%d ", c[i]);
    }
    
    // Cleanup
    free(a); free(b); free(c)
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}
